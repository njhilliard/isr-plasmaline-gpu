#include "hip/hip_runtime.h"
/*

This file reads in the binary from the defined INPUT, a binary python output array into a C
array and then runs an inplace FFT using the GPU through cufft.

****Still needs the main() segregated into supporting functions****
*/

#include <stdio.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include <errno.h>
#include <time.h>
#include "hipblas.h"

//Parameters based on input array
#define IPP 250000 // 25 MHz sample rate 10 ms IPP
#define N_RANGE_GATES 4096 // 1 microsecond range gates, we want power of two
#define RANGE_GATE_STEP 25 // 1 microsecond
#define TX_LENGTH 16384 // transmit pulse length in 25 MHz sample rate (power of two)
#define RANGE_START 500
// 

/* Kernel for complex conjugate multiplication */
__global__ void
complex_conj_mult(hipfftComplex *tx, hipfftComplex *echo, hipfftComplex *batch)
{
    unsigned int block_num        = blockIdx.x + blockIdx.y * gridDim.x;
    unsigned int thread_num       = threadIdx.x + threadIdx.y * blockDim.x;
    unsigned int threads_in_block = blockDim.x * blockDim.y;
    unsigned int idx              = threads_in_block * block_num + thread_num;

    int i = idx / TX_LENGTH;
    int j = idx % TX_LENGTH;
    int ei = j + (i + RANGE_START) * RANGE_GATE_STEP;

    batch[idx] = hipCmulf(echo[ei], tx[j]);
}


__global__ void square_and_accumulate_sum(hipfftComplex *z, float *spectrum)
{
  unsigned int block_num        = blockIdx.x + blockIdx.y * gridDim.x;
  unsigned int thread_num       = threadIdx.x + threadIdx.y * blockDim.x;
  unsigned int threads_in_block = blockDim.x * blockDim.y;
  unsigned int idx              = threads_in_block * block_num + thread_num;

  spectrum[idx] += z[idx].x*z[idx].x + z[idx].y*z[idx].y;
}


int
main() 
{
/**********************************************************************/
/**** Complex multiplication of tx with echo along the range gates ****/
/**********************************************************************/

    // hipfftComplex is single precision interleaved float
    hipfftComplex *z_tx = (hipfftComplex *)malloc(TX_LENGTH * sizeof(hipfftComplex));
    hipfftComplex *z_echo = (hipfftComplex *)malloc(IPP * sizeof(hipfftComplex));
    float *spectrum = (float *)malloc(TX_LENGTH*N_RANGE_GATES*sizeof(float));

    // initializing pointers to device memory
    hipfftComplex *d_z_tx;
    hipfftComplex *d_z_echo;
    hipfftComplex *d_z_batch;

    // allocating device memory to the above pointers
    if (hipMalloc((void **) &d_z_tx, sizeof(hipfftComplex)*TX_LENGTH) != hipSuccess) {
        fprintf(stderr, "Cuda error: Failed to allocate\n");
        exit(EXIT_FAILURE);
    }
    if (hipMalloc((void **) &d_z_echo, sizeof(hipfftComplex)*IPP) != hipSuccess) {
        fprintf(stderr, "Cuda error: Failed to allocate\n");
        exit(EXIT_FAILURE);
    }
    if (hipMalloc((void **) &d_z_batch, sizeof(hipfftComplex)*TX_LENGTH*N_RANGE_GATES) != hipSuccess)
    {
        fprintf(stderr, "Cuda error: Failed to allocate d_z_batch\n");
        exit(EXIT_FAILURE);
    }

    // setup execution parameters
    dim3 dimBlock(16,16);
    dim3 dimGrid(N_RANGE_GATES/16,TX_LENGTH/16);

/*************************************************/
/**** Complex multiplication ended, begin FFT ****/
/*************************************************/

    // initializing in-place FFT plan
    hipfftHandle plan;
    if (hipfftPlan1d(&plan, TX_LENGTH, HIPFFT_C2C, N_RANGE_GATES) != HIPFFT_SUCCESS) {
      fprintf(stderr, "CUFFT error: Plan creation failed\n");
      exit(EXIT_FAILURE);
    }

    /*// executing FFT
    if (hipfftExecC2C(plan, (hipfftComplex *)d_z_batch, (hipfftComplex *)d_z_batch, HIPFFT_FORWARD)
	    != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "CUFFT error: ExecC2C Forward failed\n");
        exit(EXIT_FAILURE);
    }

    // memory clean up
    if (hipfftDestroy(plan) != HIPFFT_SUCCESS) {
        fprintf(stderr, "Cuda error: Failed to destroy\n");
        exit(EXIT_FAILURE);
    }*/

/************************************************/
/**** In-place square; spectrum accumulation ****/
/************************************************/

    // initializing spectrum pointer
    float *d_spectrum;

    // allocating device memory
    if (hipMalloc((void **) &d_spectrum, sizeof(float)*TX_LENGTH*N_RANGE_GATES) != hipSuccess)
    {
        fprintf(stderr, "Cuda error: Failed to allocate d_spectrum\n");
        exit(EXIT_FAILURE);
    }


/**** timing the process ****/
    clock_t start, end;
    int n_reps = 1000;
    start=clock();
    for( int i=0 ; i<n_reps ; i++)
    {
      if (hipMemcpy(d_z_tx, z_tx, sizeof(hipfftComplex)*TX_LENGTH, hipMemcpyHostToDevice)
	  != hipSuccess)
      {
        fprintf(stderr, "Cuda error: Memory copy failed, HtD\n");
        exit(EXIT_FAILURE);
      }
      if (hipMemcpy(d_z_echo, z_echo, sizeof(hipfftComplex)*IPP, hipMemcpyHostToDevice)
	  != hipSuccess)
      {
        fprintf(stderr, "Cuda error: Memory copy failed, HtD\n");
        exit(EXIT_FAILURE);
      }
      // form tx*echo, assume tx is already conjugated!
      complex_conj_mult<<< dimGrid, dimBlock >>>(d_z_tx, d_z_echo, d_z_batch);
      if (hipGetLastError() != hipSuccess) {
        fprintf(stderr, "Cuda error: Failed to launch kernel\n");
        exit(EXIT_FAILURE);
      }
      if (hipfftExecC2C(plan, (hipfftComplex *)d_z_batch, (hipfftComplex *)d_z_batch, HIPFFT_FORWARD)
	      != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: ExecC2C Forward failed\n");
        exit(EXIT_FAILURE);
      }
    }
    // copy results back to device
    if (hipMemcpy(spectrum, d_spectrum, sizeof(float)*TX_LENGTH*N_RANGE_GATES, hipMemcpyDeviceToHost)
        != hipSuccess)
    {
        fprintf(stderr, "Cuda error: Memory copy failed, HtD\n");
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();
    end=clock();
    hipfftDestroy(plan);
    hipblasDestroy(handle);
    hipFree(d_z_tx);
    hipFree(d_z_echo);
    hipFree(d_z_batch);
    hipFree(d_spectrum);
    free(z_tx);
    free(z_echo);
    double dt = ((double) (end-start))/CLOCKS_PER_SEC;
    printf("Time elapsed %1.2f s / 1000 echoes %1.2f speed ratio\n", dt, ((double)n_reps*0.01)/dt );
    
    return 0;
}


//void process_echoes(float **tx, float **echo, int n_ipp, float *spectrum)
//{
//}

